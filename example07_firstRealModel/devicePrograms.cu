#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>

#include "LaunchParams.h"

using namespace osc;

namespace osc {
  
  /*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
  extern "C" __constant__ LaunchParams optixLaunchParams;

  // for this simple example, we have a single ray type
  enum { SURFACE_RAY_TYPE=0, RAY_TYPE_COUNT };
  
  static __forceinline__ __device__
  void *unpackPointer( uint32_t i0, uint32_t i1 )
  {
    const uint64_t uptr = static_cast<uint64_t>( i0 ) << 32 | i1;
    void*           ptr = reinterpret_cast<void*>( uptr ); 
    return ptr;
  }

  static __forceinline__ __device__
  void  packPointer( void* ptr, uint32_t& i0, uint32_t& i1 )
  {
    const uint64_t uptr = reinterpret_cast<uint64_t>( ptr );
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
  }

  template<typename T>
  static __forceinline__ __device__ T *getPRD()
  { 
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>( unpackPointer( u0, u1 ) );
  }
  
  //------------------------------------------------------------------------------
  // closest hit and anyhit programs for radiance-type rays.
  //
  // Note eventually we will have to create one pair of those for each
  // ray type and each geometry type we want to render; but this
  // simple example doesn't use any actual geometries yet, so we only
  // create a single, dummy, set of them (we do have to have at least
  // one group of them to set up the SBT)
  //------------------------------------------------------------------------------
  
  extern "C" __global__ void __closesthit__radiance()
  {
    /*const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // compute normal:
    const int   primID = optixGetPrimitiveIndex();
    const vec3i index  = sbtData.index[primID];
    const vec3f &A     = sbtData.vertex[index.x];
    const vec3f &B     = sbtData.vertex[index.y];
    const vec3f &C     = sbtData.vertex[index.z];
    const vec3f Ng     = normalize(cross(B-A,C-A));

    const vec3f rayDir = optixGetWorldRayDirection();
    const float cosDN  = 0.2f + .8f*fabsf(dot(rayDir,Ng));
    vec3f &prd = *(vec3f*)getPRD<vec3f>();
    //prd = cosDN * sbtData.color;
    prd = cosDN * vec3f(1*sbtData.boundary,0,1*(!sbtData.boundary));
    //prd[0] = 1*sbtData.boundary; 
    //prd[1] = 1*sbtData.boundary; 
    //prd[2] = 1; */
    }
  
  extern "C" __global__ void __anyhit__radiance()
  { 
    //printf("we in anyhit\n");
    float currentTmax = __uint_as_float(optixGetPayload_2());
    float t = optixGetRayTmax();
    //printf("is t > currentTMax %d\n", (t>currentTmax));
    //printf("t %d %f ", optixGetPrimitiveIndex(), t);
    if(t > currentTmax){
      
      //printf("getting sbt data\n");
      optixSetPayload_2(__float_as_uint(t));
      const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
      //printf("got sbt data\n");
      // compute normal:
     // printf("tri stuff\n");
      const int primID = optixGetPrimitiveIndex();
      optixSetPayload_4(primID);
      
      const vec3i index  = sbtData.index[primID];
      const vec3f &A     = sbtData.vertex[index.x];
      const vec3f &B     = sbtData.vertex[index.y];
      const vec3f &C     = sbtData.vertex[index.z];
      const vec3f N      = normalize(cross(B-A,C-A));
      const vec2i neighs = sbtData.posNegNormalSections[primID];
      const bool boundary = (neighs[0] == -1 || neighs[1] == -1);
      int firstTraceMultiplier = (optixGetPayload_3() + 1) & 1; //if first trace, multiplier is 0
      //printf("tri stuff done\n");
      Particle & p =  *(Particle*)getPRD<Particle>();
      //printf("particle\n");
      if(boundary){
        
        p.simPercent = firstTraceMultiplier * p.simPercent + t;
        p.pos += p.vel * t;
        vec3f newDir = p.vel - 2.0f*dot(p.vel, N)*N;
        p.vel = newDir;
        printf("HIT BOUNDARY\n");
        //printf("%f , %f, %f position %f , %f, %f \n", p.vel.x,p.vel.y,p.vel.z, p.pos.x,p.pos.y,p.pos.z);
        optixLaunchParams.bounced[0] = 1;
        p.section = (neighs[0] != -1) * neighs[0] + (neighs[1] != -1) * neighs[1];
        //printf("written to bounced\n");
        optixTerminateRay();
      } else {
       
        //printf("update section %d\n",p.section);
      }
      //const vec3f rayDir = optixGetWorldRayDirection();
      //const float cosDN  = 0.2f + .8f*fabsf(dot(rayDir,N));
      //vec3f &prd = *(vec3f*)getPRD<vec3f>();
      //prd = cosDN * vec3f(1*boundary,0,1*(!boundary));
    }
    //printf("irnmoring intersection\n");
    optixIgnoreIntersection();
    //printf("we done anyhit\n");
  }


  
  //------------------------------------------------------------------------------
  // miss program that gets called for any ray that did not have a
  // valid intersection
  //
  // as with the anyhit/closest hit programs, in this example we only
  // need to have _some_ dummy function to set up a valid SBT
  // ------------------------------------------------------------------------------
  
  extern "C" __global__ void __miss__radiance()
  {
    //printf("in miss\n");
    /*float currentTmax = __uint_as_float(optixGetPayload_2());
    if(currentTmax > 1e10){
      vec3f &prd = *(vec3f*)getPRD<vec3f>();
      // set to constant white as background color
      prd = vec3f(1.f);
    }*/
    //printf("we in miss\n");
    int lastPrim = optixGetPayload_4();
    if(lastPrim  != INT_MAX){
      const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
      const vec3i index  = sbtData.index[lastPrim];
      const vec3f &A     = sbtData.vertex[index.x];
      const vec3f &B     = sbtData.vertex[index.y];
      const vec3f &C     = sbtData.vertex[index.z];
      const vec3f &spare = sbtData.normals[lastPrim];
      const vec3f N      = normalize(cross(B-A,C-A));
      const vec2i neighs = sbtData.posNegNormalSections[lastPrim];
      const bool boundary = (neighs[0] == -1 || neighs[1] == -1);
      /*printf("\n%f %f %f \n", A[0],A[1],A[2]);
      printf("%f %f %f \n", B[0],B[1],B[2]);
      printf("%f %f %f \n", C[0],C[1],C[2]);
      printf("%f %f %f \n", spare[0],spare[1],spare[2]);
      printf("options %d %d \n", neighs[0], neighs[1]);*/
      Particle & p =  *(Particle*)getPRD<Particle>();
      if(!boundary){
        float dotProd = dot(p.vel,N); 
        p.section = (dotProd < 0) * neighs[1] + !(dotProd < 0) * neighs[0];
        if(dotProd == 0){
          printf("eww zero dot product");
        }
      }
    }
    
    Particle & p =  *(Particle*)getPRD<Particle>();
    int zeroIfFirstTrace = (optixGetPayload_3() + 1) & 1; 
    int oneIfFirstTrace = 1 - zeroIfFirstTrace;
    p.pos = p.pos + p.vel * oneIfFirstTrace + zeroIfFirstTrace * (1-p.simPercent) * p.vel;
    p.simPercent = 1;
    //printf("%f , %f, %f, %d \n", p.vel.x,p.vel.y,p.vel.z);
    //printf("%f , %f, %f, %d \n", p.pos.x,p.pos.y,p.pos.z, p.section);
    /*printf("%f , %f, %f, %d \n", p->pos.x,p->pos.y,p->pos.z, p->section);
    p->pos += p->vel;
    p->simPercent = 1;
    printf("%f , %f, %f \n", p->vel.x,p->vel.y,p->vel.z);
    printf("%f , %f, %f, %d \n", p->pos.x,p->pos.y,p->pos.z, p->section);*/
    //printf("we done miss\n");
  }

  //------------------------------------------------------------------------------
  // ray gen program - the actual rendering happens in here
  //------------------------------------------------------------------------------
  extern "C" __global__ void __raygen__renderFrame()
  {
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    //const int iy = optixGetLaunchIndex().y;

    //const auto &camera = optixLaunchParams.camera;

    // our per-ray data for this example. what we initialize it to
    // won't matter, since this value will be overwritten by either
    // the miss or hit program, anyway
    Particle * p = &optixLaunchParams.particles[ix];
    //printf("we in raygen %d\n", ix);
    // the values we store the PRD pointer in:
    uint32_t u0, u1;
    packPointer( p, u0, u1 );

    // normalized screen plane position, in [0,1]^2
    //const vec2f screen(vec2f(ix+.5f,iy+.5f)
    //                   / vec2f(optixLaunchParams.frame.size));
    
    // generate ray direction
    vec3f pos = p->pos;
    vec3f rayDir = p->vel;
    //printf("we in raygen\n");
    //normalize(camera.direction
    //                         + (screen.x - 0.5f) * camera.horizontal
    //                         + (screen.y - 0.5f) * camera.vertical);
    uint32_t tmaxPayload = __float_as_uint(0); //float max  as an integer
    uint32_t firstTraceFlag = (int)optixLaunchParams.firstTrace;
    uint32_t lastPrimPayload = INT_MAX;
    //printf("launbching trace\n");
    //printf("start trace: vel %f , %f, %f position %f , %f, %f \n", p->vel.x,p->vel.y,p->vel.z, p->pos.x,p->pos.y,p->pos.z);
    //printf("%f \n", rayDir.x);
    //printf("%f \n", rayDir.y);
    //printf("%f \n", rayDir.z);
    float tmax = optixLaunchParams.firstTrace * 1 + (!optixLaunchParams.firstTrace) * (1-p->simPercent);
    float eps = 5e-4;
    //printf("tmax %f  %d", tmax, (int)optixLaunchParams.firstTrace);
    optixTrace(optixLaunchParams.traversable,
               pos,
               rayDir,
               eps,    // tmin
               tmax,  // tmax
               0.0f,   // rayTime
               OptixVisibilityMask( 255 ),
               OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,//OPTIX_RAY_FLAG_NONE,
               SURFACE_RAY_TYPE,             // SBT offset
               RAY_TYPE_COUNT,               // SBT stride
               SURFACE_RAY_TYPE,             // missSBTIndex 
               u0, u1 , tmaxPayload, firstTraceFlag, lastPrimPayload);
    //printf("trace launched\n");

    //const int r = int(255.99f*pixelColorPRD.x);
    //const int g = int(255.99f*pixelColorPRD.y);
    //const int b = int(255.99f*pixelColorPRD.z);

    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    /*const uint32_t rgba = 0xff000000
      | (r<<0) | (g<<8) | (b<<16);*/

    // and write to frame buffer ...
    //const uint32_t fbIndex = ix+iy*optixLaunchParams.frame.size.x;
    //optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
  }
  
} // ::osc
